#include "hip/hip_runtime.h"
template<int BlockDim, int WARP_SZ, int DUP_REM>
__device__ __forceinline__ void BFS_BlockKernelB (	      int* __restrict__		devNode,
													      int* __restrict__		devEdge,
													   dist_t* __restrict__ 	colors,
														  int* __restrict__ 	F1,
														  int* __restrict__ 	F2,
														  int* __restrict__ 	devF2,
														  int* __restrict__		F2SizePtr,
													      int FrontierSize, int level,
													volatile long long int*		HashTable,
														  bool*                 devAdjMatrix) {

		int Queue[REG_QUEUE];
		int founds = 0;
		// for (int t = Tid >> _Log2<WARP_SZ>::VALUE; t < FrontierSize; t += BlockDim / WARP_SZ) {
		for (int t = Tid >> _Log2<WARP_SZ>::VALUE; t < FrontierSize; t += BlockDim>> _Log2<WARP_SZ>::VALUE) {
			const int index = F1[t];
			const int start = devNode[index];
			int end = devNode[index + 1];

			EdgeVisit<BlockDim, WARP_SZ, DUP_REM>(devEdge, colors, F2, F2SizePtr, start, end, Queue, founds, level, HashTable, index, devAdjMatrix);
		}

		FrontierWrite::Write<BlockDim, FrontierWrite::SIMPLE>(F2, F2SizePtr, Queue, founds);
		// int WarpPos, n, total;
		// singleblockQueueAdd(founds, F2SizePtr, WarpPos, n, total, level, (int*) &SMem[TEMP_POS]);

		// if (WarpPos + total >= BLOCK_FRONTIER_LIMIT) {
		// 	if (WarpPos < BLOCK_FRONTIER_LIMIT)
		// 		SMem[0] = WarpPos;
		// 	writeOPT<SIMPLE, STORE_DEFAULT>(devF2, Queue, founds, WarpPos, n, total);
		// } else {
		// 	writeOPT<SIMPLE, STORE_DEFAULT>(F2, Queue, founds, WarpPos, n, total);
		// }
}



// #define fun(a)		BFS_BlockKernelB<1024, (a), DUP_REM>\
// 							(devNodes, devEdges, colors, devF1Pointer, devF2Pointer, NULL, F2SizePtr, frontierSize, level, NULL, devAdjMatrix);

#define fun(a)		BFS_BlockKernelB<BLOCKDIM, (a), DUP_REM>\
							(devNodes, devEdges, colors, devF1Pointer, devF2Pointer, NULL, F2SizePtr, frontierSize, level, NULL, devAdjMatrix);

template<int DUP_REM>
__global__ void BFS_BlockKernel (		  int* __restrict__		devNodes,
										  int* __restrict__		devEdges,
									   dist_t* __restrict__ 	colors,
										  int* __restrict__ 	devF1,
										  int* __restrict__	 	devF2,
									const int                   graphVertices,
									const int                   graphEdges,
									      bool*                 devAdjMatrix,
										  int                   step) {

	__shared__ int frontierSize;
	__shared__ int visitedNodes;
	__shared__ int level;
	__shared__ int devF2SizeBlock[4];
	__shared__ int size;

	int *devF1Pointer;
	int *devF2Pointer;
	int *F2SizePtr;

	// Start a new step
	if (step == 0) {
		if (Tid < 4)
			devF2SizeBlock[Tid] = 0;

		if (Tid == BLOCKDIM - 1) {
			level = 1;
			visitedNodes = 0;
			frontierSize = 1;
		}

		devF1Pointer = &devF1[SHARED_MEMORY * blockIdx.x];
		devF2Pointer = &devF2[SHARED_MEMORY * blockIdx.x];
	}
	// Continue the step
	else {
		if (Tid < 4)
			devF2SizeBlock[Tid] = saveF2BlockSize[4 * blockIdx.x + Tid];

		if (Tid == BLOCKDIM - 1) {
			visitedNodes = 0;
			frontierSize = saveFrontier[blockIdx.x];
			level = saveLevels[blockIdx.x];
		}

		devF1Pointer = savePointerF1[blockIdx.x];
		devF2Pointer = savePointerF2[blockIdx.x];
	}

	__syncthreads();

	// Visit
	while (frontierSize && (devGlobalVisited <= (graphEdges / 100) * DEEPNESS))	{
		F2SizePtr = &devF2SizeBlock[level & 3];

		if (Tid == 0) {
			size = logValueDevice<BLOCKDIM, MIN_VW, MAX_VW>(frontierSize);
			devF2SizeBlock[(level + 1) & 3] = 0;
			visitedNodes += frontierSize;
			atomicAdd(&devGlobalVisited, frontierSize);
		}

		__syncthreads();
		def_SWITCH(size);
		__syncthreads();

		swapDev(devF1Pointer, devF2Pointer);

		if (Tid == 0) {
			level++;
			frontierSize = F2SizePtr[0];
		}

		__syncthreads();
	}

	// Save F2 status after the run
	if (Tid < 4)
		saveF2BlockSize[4 * blockIdx.x + Tid] = devF2SizeBlock[Tid];

	// Save general status after the run
	if (Tid == 5)
	{
		// Frontier
		saveFrontier[blockIdx.x] = frontierSize;
		// Level
		saveLevels[blockIdx.x] = level;

		// Pointers
		savePointerF1[blockIdx.x] = devF1Pointer;
		savePointerF2[blockIdx.x] = devF2Pointer;

		// count number of recursion in while loop per block
		// atomicAdd(&devGlobalLevel, level);

		// count number of active block in execution
		// if (visitedNodes > 0)
		// 	atomicAdd(&devActiveBlocks, 1);
	}
}

#undef fun
